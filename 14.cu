
#include <hip/hip_runtime.h>
#include <stdio.h>
// #include <cutil.h>

#define MAX 1000000
#define MAX_ITERATIONS 1000

#define CUDA_SAFE_CALL(x) x

__global__ void kernel(int* a) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  int i = 0;
  unsigned int answer = idx;

  if (idx != 0 && idx <= MAX) {
    while (answer != 1 && i < MAX_ITERATIONS) {
      if ((answer & 1) == 0) { answer = answer >> 1; }
      else { answer = 3 * answer + 1; }
      i++;
    }
  }

  if (i == MAX_ITERATIONS) {
    a[idx] = 69696969;
  } else {
    a[idx] = i;
  }
}

int main() {
  int threads_per_block = ceil(MAX / 256.0);
  printf("threads per block = %d\n", threads_per_block);
  int dimx = threads_per_block*256;
  int num_bytes = dimx * sizeof(int);

  int *d_a = 0, *h_a = 0;

  h_a = (int*)malloc(num_bytes);
  CUDA_SAFE_CALL(hipMalloc((void**)&d_a, num_bytes));

  if (0==h_a || 0==d_a) {
    printf("can't allocate memory");
  }

  CUDA_SAFE_CALL(hipMemset(d_a, 0, num_bytes));
  CUDA_SAFE_CALL(hipMemcpy(d_a, h_a, num_bytes, hipMemcpyHostToDevice));

  hipEvent_t start, stop;
  CUDA_SAFE_CALL(hipEventCreate(&start); hipEventCreate(&stop));

  CUDA_SAFE_CALL(hipEventRecord(start, 0));
  kernel<<<threads_per_block, 256>>>(d_a);
  CUDA_SAFE_CALL(hipEventRecord(stop, 0));


  CUDA_SAFE_CALL(hipEventSynchronize(stop));
  float et;
  CUDA_SAFE_CALL(hipEventElapsedTime(&et, start, stop));

  CUDA_SAFE_CALL(hipEventDestroy(start)); CUDA_SAFE_CALL(hipEventDestroy(stop));

  printf("kernel execution time: %8.6fms\n", et);

  CUDA_SAFE_CALL(hipMemcpy(h_a, d_a, num_bytes, hipMemcpyDeviceToHost));

  int max = 0;
  for(int i=0; i<dimx; i++) {
//    printf("%d ", h_a[i]);
    if (h_a[i] > max) max = h_a[i];
  }
//  printf("\n");
  printf("max is %d\n", max);

  free(h_a);
  CUDA_SAFE_CALL(hipFree(d_a));

  return 0;
} 
